#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define RANGE 11.79

/*** TODO: insert the declaration of the kernel function below this line ***/
__global__
void vecGPU(int n, float* a, float* b, float* c) {
	int i = threadIdx.x + (blockDim.x * blockIdx.x);
	if (i < n) {
		c[i] += (a[i] + b[i]);
	}
	
  
}

/**** end of the kernel declaration ***/


int main(int argc, char *argv[]){

	int n = 0; //number of elements in the arrays
	int i;  //loop index
	float *a, *b, *c; // The arrays that will be processed in the host.
	float *temp;  //array in host used in the sequential code.
	float *ad, *bd, *cd; //The arrays that will be processed in the device.
	clock_t start, end; // to meaure the time taken by a specific part of code
	
	if(argc != 2){
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
		}
		
	n = atoi(argv[1]);
	printf("Each vector will have %d elements\n", n);
	
	
	//Allocating the arrays in the host
	
	if( !(a = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}
	
	if( !(b = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array b\n");
	   exit(1);
	}
	
	if( !(c = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array c\n");
	   exit(1);
	}
	
	if( !(temp = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array temp\n");
	   exit(1);
	}
	
	//Fill out the arrays with random numbers between 0 and RANGE;
	srand((unsigned int)time(NULL));
	for (i = 0; i < n;  i++){
        a[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		temp[i] = c[i]; //temp is just another copy of C

		
	}
	
    //The sequential part
	start = clock();
	for(i = 0; i < n; i++) {
		temp[i] += a[i] * b[i];
		printf("a is %f\n",a[i]);
	}
	end = clock();
	printf("Total time taken by the sequential part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);

    /******************  The start GPU part: Do not modify anything in main() above this line  ************/
	//The GPU part
	start = clock();
	
	/* TODO: in this part you need to do the following:
		1. allocate ad, bd, and cd in the device
		2. send a, b, and c to the device
		3. write the kernel, call it: vecGPU
		4. Call the kernel (the kernel itself will be written at the comment at the end of this file), 
		   you need to write the number of threads, blocks, etc and their geometry.
		5. Bring the cd array back from the device and store it in c array (declared earlier in main)
		6. free ad, bd, and cd
	*/
	//Allocation ad,bd, and cd
	size_t arr_size = n*sizeof(float);
	hipMalloc(&ad, arr_size);
	hipMalloc(&bd, arr_size);
	hipMalloc(&cd, arr_size);
	
	//send a,b,c to device
	hipMemcpy(ad, a, arr_size, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, arr_size, hipMemcpyHostToDevice);
	hipMemcpy(cd, c, arr_size, hipMemcpyHostToDevice);
	
	//call kernel
	vecGPU<<<4,500>>>(n,ad,bd,cd);
	//bring back cd, store in c
	hipMemcpy(c,cd,arr_size, hipMemcpyDeviceToHost);
	
	//free ad,bd,cd
	hipFree(ad);
	hipFree(bd);
	hipFree(cd);



	end = clock();
	printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	/******************  The end of the GPU part: Do not modify anything in main() below this line  ************/
	
	//checking the correctness of the GPU part
	
	for(i = 0; i < n; i++) {
	  if( fabsf(temp[i] - c[i]) >= 0.009) {//compare up to the second degit in floating point 
		printf("Element %d in the result array does not match the sequential version\n", i);
	  }
	  
	}
	// Free the arrays in the host
	free(a); free(b); free(c); free(temp);

	return 0;
}


/**** TODO: Write the kernel itself below this line *****/
