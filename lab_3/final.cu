#include "hip/hip_runtime.h"
/* Author: Mark Ebersole � NVIDIA Corporation  */
/* Source: https://developer.nvidia.com/cuda-education  */

#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ printing(){
 __shared__ int x = 7;
 if(blockIdx.x > 0)
 printf("%d", blockIdx.x);
 else
 printf("x");
 __synchthreads();
 printf("%d", threadIdx.x);
} 
int main(void)
{
	
	printing<<<3,2>>>();
	hipDeviceSynchronize();

	return 0;
}